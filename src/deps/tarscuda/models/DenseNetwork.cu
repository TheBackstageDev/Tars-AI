#include "hip/hip_runtime.h"
#include "DenseNetwork.hpp"
#include <hip/hip_fp16.h>
#include <mma.h>

namespace NCUDA_NETWORK
{
    __device__ float sigmoid(float x)
    {
        return 1 / (1 + exp(-x));
    }

    // M Is rows A, K is cols A, N is cols B
    __device__ void gemm(half* A, half* B, half* output, int M, int K, int N, bool transposeA, bool transposeB)
    {

    }

    // Outputs means the vector where all summed changes will go
    __global__ void trainKernel(half* weights, half* biases, half* outWeights, half* outBiases, float learningRate) 
    {

    }

    void denseTrain(std::vector<std::vector<NTARS::DATA::TrainingData<std::vector<float>>>> trainingData, float trainingRate, NTARS::DenseNeuralNetwork& network)
    {
        float* outputs; 
    }
}
