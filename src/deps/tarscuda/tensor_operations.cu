#include "tensor_operations.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

namespace TCUDA
{
    inline void CUDA_CHECK(hipError_t cudaStatus) 
    {
        if (cudaStatus != hipSuccess) {
            throw fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
        }
    }

    inline void CUBLAS_CHECK(hipblasStatus_t hipblasStatus_t) 
    {
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
            throw fprintf(stderr, "CUBLAS error: %d\n", hipblasStatus_t);
        }
    }

/*  uses Cuda Cores  
 bool matrixMultiply(double* x, double* y, double* result, int rowsX, int colsX, int colsY) 
    {
        double *d_x, *d_y, *d_result;
        size_t sizeX = rowsX * colsX * sizeof(double);
        size_t sizeY = colsX * colsY * sizeof(double);
        size_t sizeResult = rowsX * colsY * sizeof(double);
    
        CUDA_CHECK(hipMalloc(&d_x, sizeX));
        CUDA_CHECK(hipMalloc(&d_y, sizeY));
        CUDA_CHECK(hipMalloc(&d_result, sizeResult));
    
        CUDA_CHECK(hipMemcpy(d_x, x, sizeX, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_y, y, sizeY, hipMemcpyHostToDevice));
    
        dim3 blockDim(16, 16);
        dim3 gridDim((colsY + 15) / 16, (rowsX + 15) / 16);
        matrixMultiplyKernel<<<gridDim, blockDim>>>(d_x, d_y, d_result, rowsX, colsX, colsY);

        CUDA_CHECK(hipMemcpy(result, d_result, sizeResult, hipMemcpyDeviceToHost));

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_result);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }
    
        return true; 
    } */

    void swapValue(int& val1, int& val2)
    {
        int temp = val1;
        val1 = val2;
        val2 = temp;
    }

    bool matrixMultiply(float* x, float* y, float* result, int rowsX, int colsX, int colsY, bool transposeX, bool transposeY) 
    {
        float *d_a, *d_b, *d_result;

        const float alpha = 1.0; // Scalar multiplier for matrix multiplication
        const float beta = 0.0;  // Scalar multiplier for initial values in result

        if (transposeX) {
            swapValue(rowsX, colsX);
        }
        if (transposeY) {
            swapValue(colsX, colsY);
        }

        size_t sizeX = rowsX * colsX * sizeof(float);
        size_t sizeY = colsX * colsY * sizeof(float);
        size_t sizeResult = rowsX * colsY * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_a, sizeX)); 
        CUDA_CHECK(hipMalloc(&d_b, sizeY));
        CUDA_CHECK(hipMalloc(&d_result, sizeResult));
        
        CUDA_CHECK(hipMemcpy(d_a, x, sizeX, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, y, sizeY, hipMemcpyHostToDevice));

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        hipblasOperation_t opX = transposeX ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        hipblasOperation_t opY = transposeY ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        CUBLAS_CHECK(hipblasSgemm(cublasHandle, opX, opY, colsY, rowsX, colsX, &alpha, d_b, colsY, d_a, colsX, &beta, d_result, colsY));
        CUDA_CHECK(hipMemcpy(result, d_result, sizeResult, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_b));
        CUDA_CHECK(hipFree(d_result));
        CUBLAS_CHECK(hipblasDestroy(cublasHandle));

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }

        return true;
    }

    bool matrixElementWiseMultiply(float* x, float* y, float* result, size_t size)
    {
        float *d_a, *d_b, *d_result;
    
        size_t dataSize = size * sizeof(float);
    
        CUDA_CHECK(hipMalloc(&d_a, dataSize));
        CUDA_CHECK(hipMalloc(&d_b, dataSize));
        CUDA_CHECK(hipMalloc(&d_result, dataSize));

        CUDA_CHECK(hipMemcpy(d_a, x, dataSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, y, dataSize, hipMemcpyHostToDevice));
    
        dim3 blockDim(256); 
        dim3 gridDim((size + blockDim.x - 1) / blockDim.x); 

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        CUBLAS_CHECK(hipblasSdgmm(
            cublasHandle,                          // cuBLAS handle
            HIPBLAS_SIDE_LEFT,                      // Side mode (LEFT means x is the diagonal matrix)
            size,                                  // Number of rows in the matrix
            1,                                     // Number of columns in the matrix
            d_b, size,                             // Input matrix y (column vector)
            d_a, 1,                                // Input vector x (diagonal-like input)
            d_result, size                         // Resulting output matrix
        ));

        CUDA_CHECK(hipMemcpy(result, d_result, dataSize, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_b));
        CUDA_CHECK(hipFree(d_result));
        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
    
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }
        return true;
    }
    
} // namespace TCUDA

