#include "tensor_operations.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

namespace TCUDA
{
    inline void CUDA_CHECK(hipError_t cudaStatus) 
    {
        if (cudaStatus != hipSuccess) {
            throw fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
        }
    }

    inline void CUBLAS_CHECK(hipblasStatus_t hipblasStatus_t) 
    {
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
            throw fprintf(stderr, "CUBLAS error: %d\n", hipblasStatus_t);
        }
    }

    void swapValue(int& val1, int& val2)
    {
        int temp = val1;
        val1 = val2;
        val2 = temp;
    }

    bool matrixMultiply(float* x, float* y, float* result, int rowsX, int colsX, int colsY, bool transposeX, bool transposeY) 
    {
        float *d_a, *d_b, *d_result;

        const float alpha = 1.0; // Scalar multiplier for matrix multiplication
        const float beta = 0.0;  // Scalar multiplier for initial values in result

        if (transposeX) {
            swapValue(rowsX, colsX);
        }
        if (transposeY) {
            swapValue(colsX, colsY);
        }

        size_t sizeX = rowsX * colsX * sizeof(float);
        size_t sizeY = colsX * colsY * sizeof(float);
        size_t sizeResult = rowsX * colsY * sizeof(float);

        CUDA_CHECK(hipMallocManaged(&d_a, sizeX)); 
        CUDA_CHECK(hipMallocManaged(&d_b, sizeY));
        CUDA_CHECK(hipMallocManaged(&d_result, sizeResult));
        
        CUDA_CHECK(hipMemcpy(d_a, x, sizeX, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, y, sizeY, hipMemcpyHostToDevice));

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        hipblasOperation_t opX = transposeX ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        hipblasOperation_t opY = transposeY ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        CUBLAS_CHECK(hipblasSgemm(cublasHandle, opX, opY, colsY, rowsX, colsX, &alpha, d_b, colsY, d_a, colsX, &beta, d_result, colsY));
        hipDeviceSynchronize();

        CUDA_CHECK(hipMemcpy(result, d_result, sizeResult, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_b));
        CUDA_CHECK(hipFree(d_result));
        CUBLAS_CHECK(hipblasDestroy(cublasHandle));

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }

        return true;
    }

    bool matrixElementWiseMultiply(float* x, float* y, float* result, size_t size)
    {
        float *d_a, *d_b, *d_result;
    
        size_t dataSize = size * sizeof(float);
    
        CUDA_CHECK(hipMallocManaged(&d_a, dataSize));
        CUDA_CHECK(hipMallocManaged(&d_b, dataSize));
        CUDA_CHECK(hipMallocManaged(&d_result, dataSize));

        CUDA_CHECK(hipMemcpy(d_a, x, dataSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, y, dataSize, hipMemcpyHostToDevice));
    
        dim3 blockDim(256); 
        dim3 gridDim((size + blockDim.x - 1) / blockDim.x); 

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        CUBLAS_CHECK(hipblasSdgmm(
            cublasHandle,                          // cuBLAS handle
            HIPBLAS_SIDE_LEFT,                      // Side mode (LEFT means x is the diagonal matrix)
            size,                                  // Number of rows in the matrix
            1,                                     // Number of columns in the matrix
            d_b, size,                             // Input matrix y (column vector)
            d_a, 1,                                // Input vector x (diagonal-like input)
            d_result, size                         // Resulting output matrix
        ));
        hipDeviceSynchronize();

        CUDA_CHECK(hipMemcpy(result, d_result, dataSize, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_b));
        CUDA_CHECK(hipFree(d_result));
        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
    
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }
        return true;
    }
    
} // namespace TCUDA

