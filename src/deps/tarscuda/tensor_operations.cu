#include "hip/hip_runtime.h"
#include "tensor_operations.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

namespace TCUDA
{
    inline void CUDA_CHECK(hipError_t cudaStatus) 
    {
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
            throw std::runtime_error("CUDA error");
        }
    }

    inline void CUBLAS_CHECK(hipblasStatus_t hipblasStatus_t) 
    {
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "CUBLAS error: %d\n", hipblasStatus_t);
            exit(-1);
        }
    }

/*  uses Cuda Cores   bool matrixMultiply(double* x, double* y, double* result, int rowsX, int colsX, int colsY) 
    {
        double *d_x, *d_y, *d_result;
        size_t sizeX = rowsX * colsX * sizeof(double);
        size_t sizeY = colsX * colsY * sizeof(double);
        size_t sizeResult = rowsX * colsY * sizeof(double);
    
        CUDA_CHECK(hipMalloc(&d_x, sizeX));
        CUDA_CHECK(hipMalloc(&d_y, sizeY));
        CUDA_CHECK(hipMalloc(&d_result, sizeResult));
    
        CUDA_CHECK(hipMemcpy(d_x, x, sizeX, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_y, y, sizeY, hipMemcpyHostToDevice));
    
        dim3 blockDim(16, 16);
        dim3 gridDim((colsY + 15) / 16, (rowsX + 15) / 16);
        matrixMultiplyKernel<<<gridDim, blockDim>>>(d_x, d_y, d_result, rowsX, colsX, colsY);

        CUDA_CHECK(hipMemcpy(result, d_result, sizeResult, hipMemcpyDeviceToHost));

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_result);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }
    
        return true; 
    } */
    bool matrixMultiply(float* x, float* y, float* result, int rowsX, int colsX, int colsY, bool transposeX, bool transposeY) 
    {
        float *d_a, *d_b, *d_result;

        // scalar coefficients
        const float alpha = 1.0;
        const float beta = 0.0;

        size_t sizeX = rowsX * colsX * sizeof(float);
        size_t sizeY = colsX * colsY * sizeof(float);
        size_t sizeResult = rowsX * colsY * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_a, sizeX)); 
        CUDA_CHECK(hipMalloc(&d_b, sizeY));
        CUDA_CHECK(hipMalloc(&d_result, sizeResult));
        
        CUDA_CHECK(hipMemcpy(d_a, x, sizeX, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, y, sizeY, hipMemcpyHostToDevice));

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        hipblasOperation_t opX = transposeX ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        hipblasOperation_t opY = transposeY ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        CUBLAS_CHECK(hipblasSgemm(cublasHandle, opX, opY, colsY, rowsX, colsX, &alpha, d_b, colsY, d_a, colsX, &beta, d_result, colsY));
        CUDA_CHECK(hipMemcpy(result, d_result, sizeResult, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_b));
        CUDA_CHECK(hipFree(d_result));
        CUBLAS_CHECK(hipblasDestroy(cublasHandle));

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }

        return true;
    }

    bool matrixMultiply(std::vector<std::vector<float>>& matrixA, std::vector<std::vector<float>>& matrixB,  std::vector<std::vector<float>>* resultMatrix, bool transposeX, bool transposeY)
    {
        const float alpha = 1.0;
        const float beta = 0.0;

        size_t sizeX = matrixA.size() * matrixA[0].size() * sizeof(float);
        size_t sizeY =  matrixA[0].size() * matrixB[0].size() * sizeof(float);
        size_t sizeResult = matrixA.size() * matrixB[0].size() * sizeof(float);

        return false;
    }

    __global__ void matrixElementWiseMultiplyKernel(const float* x, const float* y, float* result, size_t size)
    {
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            result[idx] = x[idx] * y[idx];
        }
    }
    
    bool matrixElementWiseMultiply(float* x, float* y, float* result, size_t size)
    {
        float *d_x, *d_y, *d_result;
    
        size_t dataSize = size * sizeof(float);
    
        CUDA_CHECK(hipMalloc(&d_x, dataSize));
        CUDA_CHECK(hipMalloc(&d_y, dataSize));
        CUDA_CHECK(hipMalloc(&d_result, dataSize));

        CUDA_CHECK(hipMemcpy(d_x, x, dataSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_y, y, dataSize, hipMemcpyHostToDevice));
    
        dim3 blockDim(256); 
        dim3 gridDim((size + blockDim.x - 1) / blockDim.x); 

        matrixElementWiseMultiplyKernel<<<gridDim, blockDim>>>(d_x, d_y, d_result, size);

        CUDA_CHECK(hipMemcpy(result, d_result, dataSize, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_x));
        CUDA_CHECK(hipFree(d_y));
        CUDA_CHECK(hipFree(d_result));
    
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            return false;
        }
        return true;
    }
    
} // namespace TCUDA

